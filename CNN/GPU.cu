#include "GPU.cuh"

hipStream_t GPU::createStream()
{
    hipStream_t out;
    check(hipStreamCreate(&out));
    return out;
}
void GPU::destroyStream(hipStream_t stream) {check(hipStreamDestroy(stream));}

void GPU::destroyHostPinned(void *arr) {check(hipHostFree(arr));}

void GPU::destroyDeviceMem(void *arr,hipStream_t stream) {check(hipFreeAsync(arr,stream));}

void GPU::sync() {check(hipDeviceSynchronize());}
void GPU::sync(hipStream_t stream) {check(hipStreamSynchronize(stream));}