#include "hip/hip_runtime.h"
#include "pool_test.cuh"
#include "pool.cuh"

constexpr const unsigned Ir = 6,Ic = 6,Ich = 2,
                         Or = 2,Oc = 3,Och = 1,
                         Pr = 3,Pc = 2,Pch = 2;

constexpr const double INI_I[Ir*Ic*Ich] = { 0,35,   4,31,   8,27,  12,23,  16,19,   1,34,
                                           19,16,  20,15,  24,11,  28, 7,  21,14,   5,30,
                                           15,20,  31, 4,  32, 3,  33, 2,  25,10,   9,26,
                                           11,24,  27, 8,  35, 0,  34, 1,  29, 6,  13,22,
                                            7,28,  23,12,  30, 5,  26, 9,  22,13,  17,18,
                                            3,32,  18,17,  14,21,  10,25,   6,29,   2,33};
constexpr const double INI_dO[Or*Oc*Och] = {-3,-2,-1,
                                             0, 1, 2};

constexpr const double EXP_O[Or*Oc*Och] = {35,33,34,
                                           32,35,33};
constexpr const double EXP_dI[Ir*Ic*Ich] = {0,-3, 0,0, 0,0,  0,0, 0,0, 0,-1,
                                            0, 0, 0,0, 0,0,  0,0, 0,0, 0, 0,
                                            0, 0, 0,0, 0,0, -2,0, 0,0, 0, 0,
                                            0, 0, 0,0, 1,0,  0,0, 0,0, 0, 0,
                                            0, 0, 0,0, 0,0,  0,0, 0,0, 0, 0,
                                            0, 0, 0,0, 0,0,  0,0, 0,0, 0, 2};

#define CPY(size,dst,src) \
for(unsigned i = 0;i < size;++i)\
    dst[i] = src[i]

#define CMP1D(name,X,exp,res) \
{\
    printf(name ":\n");\
    hasErr = 0;\
    for(unsigned x = 0;x < X;++x)\
    {\
        if(exp[x] != res[x])\
        {\
            hasErr = 1;\
            printf(\
                "\t<%u> %f != %f\n",\
                x,exp[x],res[x]\
            );\
        }\
    }\
    if(!hasErr) printf("\tNo Errors.\n");\
}

#define CMP3D(name,X,Y,Z,exp,res) \
{\
    printf(name ":\n");\
    hasErr = 0;\
    for(unsigned x = 0;x < X;++x)\
    {\
        for(unsigned y = 0;y < Y;++y)\
        {\
            for(unsigned z = 0;z < Z;++z)\
            {\
                unsigned idx = x*Y*Z+y*Z+z;\
                if(exp[idx] != res[idx])\
                {\
                    hasErr = 1;\
                    printf(\
                        "\t<%u,%u,%u> %f != %f\n",\
                        x,y,z,exp[idx],res[idx]\
                    );\
                }\
            }\
        }\
    }\
    if(!hasErr) printf("\tNo Errors.\n");\
}

void pool_test::testSTC()
{
    bool hasErr;
    double *O = (double*)malloc(sizeof(double)*Or*Oc*Och),*dO;
    pool::STCLayer<double> layer(&O,&dO,Ir,Ic,Ich,Or,Oc,Och,Pr,Pc,Pch);
    CPY(Ir*Ic*Ich,layer.I,INI_I);

    layer.forward();
    CMP3D("O",Or,Oc,Och,EXP_O,O);
    free(O);
    if(hasErr) return;

    dO = (double*)malloc(sizeof(double)*Or*Oc*Och);
    CPY(Or*Oc*Och,dO,INI_dO);

    layer.backward();
    CMP3D("dI",Ir,Ic,Ich,EXP_dI,layer.dI);
}
void pool_test::testOMP()
{
    bool hasErr;
    double *O = (double*)malloc(sizeof(double)*Or*Oc*Och),*dO;
    pool::OMPLayer<double> layer(&O,&dO,Ir,Ic,Ich,Or,Oc,Och,Pr,Pc,Pch);
    CPY(Ir*Ic*Ich,layer.I,INI_I);

    layer.forward();
    CMP3D("O",Or,Oc,Och,EXP_O,O);
    free(O);
    if(hasErr) return;

    dO = (double*)malloc(sizeof(double)*Or*Oc*Och);
    CPY(Or*Oc*Och,dO,INI_dO);

    layer.backward();
    CMP3D("dI",Ir,Ic,Ich,EXP_dI,layer.dI);
}
void pool_test::testGPU()
{
    hipStream_t stream = GPU::createStream();

    bool hasErr;
    double *O,*dO,*h_O,*h_dI;

    pool::GPULayer<double> layer(&O,&dO,&h_O,Ir,Ic,Ich,Or,Oc,Och,Pr,Pc,Pch);

    GPU::allocHostPinned(&h_O,Or*Oc*Och);
    GPU::allocDeviceMem(&layer.I,Ir*Ic*Ich,stream);
    GPU::transfer<double,hipMemcpyHostToDevice>((double*)INI_I,layer.I,Ir*Ic*Ich,stream);
    layer.forward(stream);
    GPU::transfer<double,hipMemcpyDeviceToHost>(O,h_O,Or*Oc*Och,stream);
    GPU::destroyDeviceMem(O,stream);
    GPU::sync(stream);
    CMP3D("O",Or,Oc,Och,EXP_O,h_O);
    GPU::destroyHostPinned(h_O);
    if(hasErr)
    {
        GPU::sync(stream);
        GPU::destroyStream(stream);
        GPU::sync();
        return;
    }
    
    GPU::allocHostPinned(&h_dI,Ir*Ic*Ich);
    GPU::allocDeviceMem(&dO,Or*Oc*Och,stream);
    GPU::transfer<double,hipMemcpyHostToDevice>((double*)INI_dO,dO,Or*Oc*Och,stream);
    layer.backward(stream);
    GPU::transfer<double,hipMemcpyDeviceToHost>(layer.dI,h_dI,Ir*Ic*Ich,stream);
    GPU::destroyDeviceMem(layer.dI,stream);
    GPU::sync(stream);
    CMP3D("dI",Ir,Ic,Ich,EXP_dI,h_dI);
    GPU::destroyHostPinned(h_dI);

    GPU::sync(stream);
    GPU::destroyStream(stream);
    GPU::sync();
}